#include "hip/hip_runtime.h"
// src/rgb2yuv.cu
#include "rgb2yuv.cuh"
#include <hip/hip_runtime.h>
//不需要使用模板，因为一般的嵌入式开发板只支持fp32
// 计算Y分量的核函数
__global__ void rgb2yuv_y_kernel(const float* __restrict__ rgb,
                                 float* __restrict__ y,
                                 int width, int height) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y_coord = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y_coord >= height) return;

    const int R = y_coord * width + x;
    const int G = width * height + y_coord * width + x;
    const int B = 2 * width * height + y_coord * width + x;

    const float fr = rgb[R];
    const float fg = rgb[G];
    const float fb = rgb[B];

    y[y_coord * width + x] = 0.299f * fr + 0.587f * fg + 0.114f * fb;
}

// 计算UV分量的核函数（下采样到420）
__global__ void rgb2yuv_uv_kernel(const float* __restrict__ rgb,
                                  float* __restrict__ yuv,
                                  int width, int height) {
    const int u_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int u_y = blockIdx.y * blockDim.y + threadIdx.y;

    const int uv_width = width / 2;
    const int uv_height = height / 2;

    if (u_x >= uv_width || u_y >= uv_height) return;

    // 原图起始位置
    const int orig_x = u_x * 2;
    const int orig_y = u_y * 2;

    float u_sum = 0, v_sum = 0;
    int count = 0;

    // 遍历2x2块
    for (int dy = 0; dy < 2; ++dy) {
        for (int dx = 0; dx < 2; ++dx) {
            const int x = orig_x + dx;
            const int y = orig_y + dy;
            
            if (x >= width || y >= height) continue;

            const int R = y * width + x;
            const int G = width * height + y * width + x;
            const int B = 2 * width * height + y * width + x;

            const float fr = rgb[R];
            const float fg = rgb[G];
            const float fb = rgb[B];

            // 计算U和V值
            u_sum += -0.169f * fr - 0.331f * fg + 0.5f * fb + 128;
            v_sum += 0.5f * fr - 0.419f * fg - 0.081f * fb + 128;
            count++;
        }
    }

    // 计算平均值
    const float u_avg = u_sum / count;
    const float v_avg = v_sum / count;

    // YUV420P内存布局：Y + U + V
    const int y_size = width * height;
    const int uv_size = uv_width * uv_height;
    const int u_index = y_size + u_y * uv_width + u_x;
    const int v_index = y_size + uv_size + u_y * uv_width + u_x;

    yuv[u_index] = u_avg;
    yuv[v_index] = v_avg;
}

void launch_rgb2yuv(Tensor* d_rgb, Tensor* d_yuv, int width, int height) {
    // 新增类型和设备断言
    assert(d_rgb->dtype() == DataType::FLOAT32 && "Input tensor must be FLOAT32");
    assert(d_yuv->dtype() == DataType::FLOAT32 && "Output tensor must be FLOAT32");
    assert(d_rgb->device() == DeviceType::GPU && "Input tensor must be on GPU");
    assert(d_yuv->device() == DeviceType::GPU && "Output tensor must be on GPU");

    // 显式获取浮点指针
    float* rgb_data = static_cast<float*>(d_rgb->data());
    float* yuv_data = static_cast<float*>(d_yuv->data());

    // 处理Y分量
    dim3 block(32, 8);
    dim3 grid((width + block.x - 1)/block.x, 
             (height + block.y - 1)/block.y);
    rgb2yuv_y_kernel<<<grid, block>>>(rgb_data, yuv_data, width, height);

    // 处理UV分量
    const int uv_width = width / 2;
    const int uv_height = height / 2;
    dim3 uv_block(16, 8);
    dim3 uv_grid((uv_width + uv_block.x - 1)/uv_block.x,
                (uv_height + uv_block.y - 1)/uv_block.y);
    rgb2yuv_uv_kernel<<<uv_grid, uv_block>>>(rgb_data, yuv_data, width, height);

    hipDeviceSynchronize();
}
